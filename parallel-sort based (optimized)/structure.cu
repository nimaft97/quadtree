#include "hip/hip_runtime.h"
#include <algorithm>
#include <iterator>

#include "structure.cuh"

__host__ __device__
void Node::initialize(int x1, int y1, int x2, int y2, int left, int right){
    initialized = true;
    setBottomLeftPoint(x1, y1);
    setTopRightPoint(x2, y2);
    setIdx(left, right);
}

__host__ __device__
void Node::setIdx(int left, int right){
    idx_s = left;
    idx_e = right;
}

__host__ __device__
bool Node::isInitialized(){
    return initialized;
}

__host__ __device__
bool Node::mustSplit(int K_MAX){
        return (idx_e - idx_s + 1) > K_MAX;
}

__host__ __device__
void Node::getIdx(int& left, int& right){
    left = idx_s;
    right = idx_e;
}

__host__ __device__
void Node::getBottomLeftPoint(int& x, int& y){
    x = x_min;
    y = y_min;
}

__host__ __device__
void Node::getTopRightPoint(int& x, int& y){
    x = x_max;
    y = y_max;
}

__host__ __device__
void Node::setChildren(int* childNodesArr){
    for (int i=0; i<4; i++)
        children[i] = childNodesArr[i];
}

__host__ __device__
void Node::setBottomLeftPoint(int x, int y){

    x_min = x;
    y_min = y;
}

__host__ __device__
void Node::setTopRightPoint(int x, int y){

    x_max = x;
    y_max = y;
}

__host__ __device__
void Node::getChildren(int* arr){
    for (int i=0; i<4; i++)
        arr[i] = children[i];
}   